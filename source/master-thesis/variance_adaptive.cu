
#include <hip/hip_runtime.h>
//#include <optixu/optixu_math_namespace.h>
////#include "optixPathTracer.h"
//
//using namespace optix;
//
//// Adaptive post processing variables and buffers
//
//rtDeclareVariable(unsigned int, window_size, , );
////rtDeclareVariable(unsigned int, max_ray_budget_total, , ) = static_cast<uint>(50u);
//rtDeclareVariable(unsigned int, max_per_frame_samples_budget, , ) = static_cast<uint>(5u);		/* this variable can be written by the user */
//
////
//// Adaptive version of pathtracing begin
////
//
///*--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------*/
///* Adaptive additional rays variables */
//rtBuffer<int4, 2>	  adaptive_samples_budget_buffer;									/* this buffer will be initialized by the host, but must also be modified by the graphics device */
//																						//rtBuffer<int4, 2>	  hoelder_refinement_buffer;										/* this buffer contains the information, where refinement will take place according to
//																						//hoelder regularity criterion, everywhere where refinement is needed value is 1, else zero */
//
//// For debug!
//rtBuffer<float4, 2>	  per_window_variance_buffer_output;
//
//rtBuffer<float4, 2>   input_buffer;														/* this buffer contains the initially rendered picture to be post processed */
//
////rtBuffer<float4, 2>   post_process_output_buffer;										/* this buffer contains the result, processed with additional adaptive rays */
//
//static __device__ __inline__ float compute_window_variance(uint2 center, uint window_size)
//{
//	uint2 screen = make_uint2(input_buffer.size().x, input_buffer.size().y);
//
//	float mean = 0.f;
//	float variance = 0.f;
//	if (per_window_variance_buffer_output[center].x < 0.0f)
//	{
//		uint squared_window_size = window_size * window_size;
//		uint half_window_size = (window_size / 2) + (window_size % 2);
//		uint2 top_left_window_corner = make_uint2(center.x - half_window_size, center.y - half_window_size);
//
//		//rtPrintf("Top left window corner: [ %d , %d ]\n\n", top_left_window_corner.x, top_left_window_corner.y);
//		//post_process_output_buffer[center] = make_float4(100.0f, 0.0f, 100.0f, 1.0f);
//
//		/* compute mean value */
//		for (uint i = 0; i < squared_window_size; i++)
//		{
//			uint2 idx = make_uint2((i % window_size + top_left_window_corner.x) % screen.x, (i / window_size + top_left_window_corner.y) % screen.y);
//			float3 input_buffer_val = make_float3(input_buffer[idx].x, input_buffer[idx].y, input_buffer[idx].z);
//			mean += 1.f / 3.f * (input_buffer_val.x + input_buffer_val.y + input_buffer_val.z);
//			//if (i % window_size <= i / window_size)
//			//{
//			//	post_process_output_buffer[idx] = make_float4(100.0f, 0.0f, 100.0f, 1.0f);
//			//}
//			//if (i % window_size == 0 && i / window_size == 0)
//			//{
//			//	//rtPrintf("Left lower corner, with window size: %d!!! \n\n", window_size);
//			//	//rtPrintf("Center: [ %d , %d ], Current global window index: [ %d , %d ] \n\n", center.x, center.y, idx.x, idx.y);
//			//	post_process_output_buffer[idx] = make_float4(100.0f, 0.0f, 100.0f, 1.0f);
//			//}
//			//if (i % window_size == window_size - 1 && i / window_size == 0)
//			//{
//			//	post_process_output_buffer[idx] = make_float4(100.0f, 0.0f, 100.0f, 1.0f);
//			//}
//			//if (i % window_size == 0 && i / window_size == window_size - 1)
//			//{
//			//	post_process_output_buffer[idx] = make_float4(100.0f, 0.0f, 100.0f, 1.0f);
//			//}
//			//if (i % window_size == window_size - 1 && i / window_size == window_size - 1)
//			//{
//			//post_process_output_buffer[idx] = make_float4(100.0f, 0.0f, 100.0f, 1.0f);
//			//post_process_output_buffer[top_left_window_corner] = make_float4(100.0f, 0.0f, 0.0f, 1.0f);
//			//}
//		}
//
//		/*mean *= 1.f/ squared_window_size;*/
//		mean = 1.f / squared_window_size * mean;
//
//		/* compute variance */
//		for (uint i = 0; i < squared_window_size; i++)
//		{
//			uint2 idx = make_uint2((i % window_size + top_left_window_corner.x) % screen.x, (i / window_size + top_left_window_corner.y) % screen.x);
//			float3 input_buffer_val = make_float3(input_buffer[idx].x, input_buffer[idx].y, input_buffer[idx].z);
//			float var = 1.f / 3.f * (input_buffer_val.x + input_buffer_val.y + input_buffer_val.z);
//			/*variance += var * var;*/
//			variance += (var * var - 2.0f * mean * var + mean * mean);
//		}
//
//		//variance = 1.f / squared_window_size * (variance) - (mean * mean);
//		variance = 1.f / squared_window_size * variance;
//
//		per_window_variance_buffer_output[center] = make_float4(variance);
//		//atomicExch(&per_window_variance_buffer_output[center].x, variance);
//
//		rtPrintf("Set variance!!!\n\n");
//	}
//	else
//	{
//		rtPrintf("Reuse variance!!!\n\n");
//		variance = per_window_variance_buffer_output[center].x;
//	}
//
//	return variance;
//};
//
//static __device__ __inline__ uint compute_samples_number(uint2 current_launch_index, float variance)
//{
//	uint samples_number = 0;
//
//	if (adaptive_samples_budget_buffer[current_launch_index].x > 0)
//	{
//		samples_number = static_cast<uint>(clamp(static_cast<float>(variance * max_per_frame_samples_budget), 0.0f, static_cast<float>(max_per_frame_samples_budget)));
//		adaptive_samples_budget_buffer[current_launch_index] = make_int4(adaptive_samples_budget_buffer[current_launch_index].x - static_cast<int>(samples_number));
//	}
//
//	return samples_number;
//};
//
//static __device__ __inline__ uint compute_current_samples_number(uint2 current_launch_index, uint window_size)
//{
//	uint sample_number = 0;
//
//	//uint additional_samples_number = 0;
//
//	size_t2 screen = input_buffer.size();
//
//	uint times_width = screen.x / window_size;
//	uint times_height = screen.y / window_size;
//
//	uint horizontal_padding = static_cast<uint>(0.5f * (screen.x - (times_width * window_size)));
//	uint vertical_padding = static_cast<uint>(0.5f * (screen.y - (times_height * window_size)));
//
//	uint half_window_size = (window_size / 2) + (window_size % 2);
//
//	uint2 times_launch_index = make_uint2(((current_launch_index.x / window_size) * window_size) % screen.x, ((current_launch_index.y / window_size) * window_size) % screen.y);
//
//	uint2 current_window_center = make_uint2(times_launch_index.x + horizontal_padding + half_window_size, times_launch_index.y + vertical_padding + half_window_size);
//
//	float variance = compute_window_variance(current_window_center, window_size);
//
//	sample_number = compute_samples_number(current_launch_index, (30.0f * variance));
//
//	return sample_number;
//};