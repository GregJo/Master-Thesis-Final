#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2017 NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optixu/optixu_math_namespace.h>
#include "optixPathTracer.h"

#include "random.h"
#include "Buffers.h"
#include "AdaptiveUtil.h"
#include "LevelHoelderAdaptive.h"
#include "MitchellFilterDevice.h"

using namespace optix;

struct PerRayData_pathtrace
{
    float3 result;
    float3 radiance;
    float3 attenuation;
    float3 origin;
    float3 direction;
    unsigned int seed;
    int depth;
    int countEmitted;
    int done;
	//int isAdaptive;
	int missed;
	int obj_id;
};

struct PerRayData_pathtrace_shadow
{
    bool inShadow;
};

// Scene wide variables
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(float,		 far_plane, , );
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(uint2,         launch_index, rtLaunchIndex, );

rtDeclareVariable(PerRayData_pathtrace, current_prd, rtPayload, );



//-----------------------------------------------------------------------------
//
//  Camera program -- main ray tracing loop
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(unsigned int,  frame_number, , );
//rtDeclareVariable(unsigned int,	 max_per_frame_samples_budget, , ) = static_cast<uint>(4u);
//rtDeclareVariable(unsigned int,  sqrt_num_samples, , );
rtDeclareVariable(unsigned int,  num_samples, , );
rtDeclareVariable(unsigned int,  rr_begin_depth, , );
rtDeclareVariable(unsigned int,  pathtrace_ray_type, , );
rtDeclareVariable(unsigned int,  pathtrace_shadow_ray_type, , );
rtDeclareVariable(float3,		 bg_color, , );

rtBuffer<ParallelogramLight>     lights;

// Adaptive post processing variables and buffers
//rtBuffer<int4, 2>				 additional_rays_buffer_input;										/* this buffer will be initialized by the host, but must also be modified by the graphics device */

//rtDeclareVariable(unsigned int, window_size, , );
rtDeclareVariable(unsigned int, max_ray_budget_total, , ) = static_cast<uint>(5u);				/* this variable will be written by the user */
rtDeclareVariable(unsigned int, max_per_launch_idx_ray_budget, , ) = static_cast<uint>(5u);		/* this variable will be written by the user */
//rtDeclareVariable(int, camera_changed, , );

static __device__ __inline__ void reset_current_total_rays_buffer(uint2 current_launch_index)
{
	output_current_total_rays_buffer[current_launch_index] = make_int4(static_cast<int>(0));
};

static __device__ __inline__ uint2 compute_variance_window_center(uint2 current_launch_index, uint window_size)
{
	size_t2 screen = output_buffer.size();

	uint times_width = screen.x / window_size;
	uint times_height = screen.y / window_size;

	uint horizontal_padding = static_cast<uint>((screen.x - (times_width * window_size)) / 2);
	uint vertical_padding = static_cast<uint>((screen.y - (times_height * window_size)) / 2);

	uint half_window_size = (window_size / 2) + (window_size % 2);

	uint2 times_launch_index = make_uint2(((current_launch_index.x / window_size) * window_size) % screen.x, ((current_launch_index.y / window_size) * window_size) % screen.y);

	uint2 current_window_center = make_uint2(times_launch_index.x + horizontal_padding + half_window_size, times_launch_index.y + vertical_padding + half_window_size);

	return current_window_center;
};


rtDeclareVariable(unsigned int, current_level_window_size, , );
rtDeclareVariable(int, next_level_begin, , ) = 0;

static __device__ void fill_buffers()
{
	uint2 current_window_centre = get_current_window_centre(launch_index, current_level_window_size);

	output_buffer[launch_index] = output_buffer[current_window_centre];
	output_scene_depth_buffer[launch_index] = output_scene_depth_buffer[current_window_centre];
	output_filter_sum_buffer[launch_index] = output_filter_sum_buffer[current_window_centre];
	output_filter_x_sample_sum_buffer[launch_index] = output_filter_x_sample_sum_buffer[current_window_centre];

	output_current_total_rays_buffer[launch_index] = output_current_total_rays_buffer[current_window_centre];

#ifdef DEBUG_HOELDER
	depth_gradient_buffer[launch_index] = depth_gradient_buffer[current_window_centre];
#endif //DEBUG_HOELDER

	object_ids_buffer[launch_index] = object_ids_buffer[current_window_centre];
};

RT_PROGRAM void pathtrace_camera()
{

	uint2 current_window_centre = get_current_window_centre(launch_index, current_level_window_size);

	size_t2 screen = output_buffer.size();

	if (frame_number == 1 || camera_changed == 1 || next_level_begin == 1/*change to something like: first_adaptive_level == true*/)
	{
		initialize_hoelder_adaptive_buffers(current_window_centre);
		initialize_hoelder_adaptive_buffers(launch_index);
	}
	if (next_level_begin == 1 && frame_number > 1)
	{
		//rtPrintf("Next level begin!\n");
		if (current_level_window_size < screen.x && current_level_window_size < screen.y /*&& get_do_refine(current_window_centre) == 1*/)
		{
			hoelder_compute_current_level_samples_count(launch_index, current_window_centre, current_level_window_size);
		}
	}
	int do_refine = get_do_refine(current_window_centre);
	///*int */do_refine = 1;

	float2 inv_screen = 1.0f / make_float2(screen) * 2.f;
	float2 centre_pixel = (make_float2(current_window_centre)) * inv_screen - 1.f;

	unsigned int samples_per_pixel = min(num_samples, max_per_frame_samples_budget);
	float3 result = make_float3(0.0f);

	unsigned int current_sqrt_num_samples = static_cast<unsigned int>(sqrtf(static_cast<float>(samples_per_pixel)));

	if (!current_sqrt_num_samples)
	{
		++current_sqrt_num_samples;
	}

	float2 jitter_scale = inv_screen / current_sqrt_num_samples;

	unsigned int seed = tea<16>(screen.x*current_window_centre.y + current_window_centre.x, frame_number);

	// Make this a clear function.
	if (camera_changed == 1 || frame_number == 1)
	{
		//rtPrintf("Reset additional rays buffer!!!\n\n");
		output_scene_depth_buffer[launch_index] = make_float4(1.0f, 1.0f, 1.0f, 1.0f);
		output_filter_sum_buffer[launch_index] = make_float4(0.0f, 0.0f, 0.0f, 1.0f);
		output_filter_x_sample_sum_buffer[launch_index] = make_float4(0.0f, 0.0f, 0.0f, 1.0f);
		reset_current_total_rays_buffer(launch_index);
		output_buffer[launch_index] = make_float4(bg_color, 1.0f);
	}

	int num_not_missed_rays = samples_per_pixel;
	if (current_window_centre.x == launch_index.x && current_window_centre.y == launch_index.y && samples_per_pixel > 0 && do_refine == 1)
	{
		do
		{
			//
			// Sample pixel using jittering
			//
			unsigned int x = samples_per_pixel % current_sqrt_num_samples;
			unsigned int y = samples_per_pixel / current_sqrt_num_samples;
			float2 jitter = make_float2(x - rnd(seed), y - rnd(seed));
			float2 d = centre_pixel + jitter*jitter_scale;
			float3 ray_origin = eye;
			float3 ray_direction = normalize(d.x*U + d.y*V + W);

			// Initialze per-ray data
			PerRayData_pathtrace prd;
			prd.result = make_float3(0.f);
			prd.attenuation = make_float3(1.f);
			prd.countEmitted = true;
			prd.done = false;
			prd.seed = seed;
			prd.depth = 0;
			prd.missed = 0;
			prd.obj_id = -1;

			// Each iteration is a segment of the ray path.  The closest hit will
			// return new segments to be traced here.
			for (;;)
			{
				if (prd.depth == 1)
				{
					float ray_length = fabsf(length((prd.origin - eye)));
					float normalized_ray_length = ray_length / far_plane;//2500.0f;

					float a = 0.0f;
					float3 old_depth = make_float3(output_scene_depth_buffer[current_window_centre]);
					output_scene_depth_buffer[current_window_centre] = make_float4(normalized_ray_length);

					object_ids_buffer[current_window_centre] = prd.obj_id;
				}

				Ray ray = make_Ray(ray_origin, ray_direction, pathtrace_ray_type, scene_epsilon, RT_DEFAULT_MAX);
				rtTrace(top_object, ray, prd);

				if (prd.done)
				{
					if (prd.missed && prd.depth == 0)
					{
						num_not_missed_rays--;
					}
					// We have hit the background or a luminaire
					prd.result += prd.radiance * prd.attenuation;
					break;
				}

				// Russian roulette termination 
				if (prd.depth >= rr_begin_depth)
				{
					float pcont = fmaxf(prd.attenuation);
					if (rnd(prd.seed) >= pcont)
						break;
					prd.attenuation /= pcont;
				}

				prd.depth++;
				prd.result += prd.radiance * prd.attenuation;

				// Update ray data for the next path segment
				ray_origin = prd.origin;
				ray_direction = prd.direction;
			}

			result += prd.result;
			seed = prd.seed;
			output_current_total_rays_buffer[current_window_centre].x++;
			float2 sample_position = make_float2(current_window_centre.x + jitter.x, current_window_centre.y + jitter.y);
			int current_total_rays = output_current_total_rays_buffer[current_window_centre].x;
			computeLevelMitchellFilterSampleContributionInNeighborhood(current_level_window_size, sample_position, current_window_centre, prd.result, screen, current_total_rays, &output_filter_sum_buffer, &output_filter_x_sample_sum_buffer);
		} while (--samples_per_pixel);


		if (num_not_missed_rays > 0)
		{
			evaluatePixelFileringEquation(current_window_centre, &output_buffer, &output_filter_sum_buffer, &output_filter_x_sample_sum_buffer);
		}
	}
	fill_buffers();
}

//-----------------------------------------------------------------------------
//
//  Emissive surface closest-hit
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3,        emission_color, , );

RT_PROGRAM void diffuseEmitter()
{
    current_prd.radiance = current_prd.countEmitted ? emission_color : make_float3(0.f);
    current_prd.done = true;
}


//-----------------------------------------------------------------------------
//
//  Lambertian surface closest-hit
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3,     diffuse_color, , );

//
// Diffuse texture and sampler
//
rtTextureSampler<float4, 2> Kd_map;
//rtTextureSampler<float4, 2> Ks_map;		// specular
rtTextureSampler<float4, 2> D_map;		// alpha texture
rtDeclareVariable(float3, texcoord, attribute texcoord, );

rtDeclareVariable(int,		  obj_id,			attribute obj_id, );

rtDeclareVariable(float3,     geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3,     shading_normal,   attribute shading_normal, );
rtDeclareVariable(optix::Ray, ray,              rtCurrentRay, );
rtDeclareVariable(float,      t_hit,            rtIntersectionDistance, );

RT_PROGRAM void diffuseTextured()
{
	float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);

	float3 hitpoint = ray.origin + t_hit * ray.direction;

	//
	// Generate a reflection ray.  This will be traced back in ray-gen.
	//
	current_prd.origin = hitpoint;

	float z1 = rnd(current_prd.seed);
	float z2 = rnd(current_prd.seed);
	float3 p;
	cosine_sample_hemisphere(z1, z2, p);
	optix::Onb onb(ffnormal);
	onb.inverse_transform(p);
	current_prd.direction = p;

	// Diffuse texture value
	const float3 diffuse_tex_sample = make_float3(tex2D(Kd_map, texcoord.x, texcoord.y));
	const float3 alpha_tex_sample = make_float3(tex2D(D_map, texcoord.x, texcoord.y));

	current_prd.attenuation = current_prd.attenuation * diffuse_tex_sample;
	current_prd.countEmitted = false;
	current_prd.obj_id = obj_id;
		
	//
	// Next event estimation (compute direct lighting).
	//
	unsigned int num_lights = lights.size();
	float3 result = make_float3(0.0f);

	for (int i = 0; i < num_lights; ++i)
	{
		// Choose random point on light
		ParallelogramLight light = lights[i];
		const float z1 = rnd(current_prd.seed);
		const float z2 = rnd(current_prd.seed);
		const float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;

		// Calculate properties of light sample (for area based pdf)
		const float  Ldist = length(light_pos - hitpoint);
		const float3 L = normalize(light_pos - hitpoint);
		const float  nDl = dot(ffnormal, L);
		const float  LnDl = dot(light.normal, L);

		// cast shadow ray
		if (nDl > 0.0f && LnDl > 0.0f)
		{
			PerRayData_pathtrace_shadow shadow_prd;
			shadow_prd.inShadow = false;
			// Note: bias both ends of the shadow ray, in case the light is also present as geometry in the scene.
			Ray shadow_ray = make_Ray(hitpoint, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist - scene_epsilon);
			rtTrace(top_object, shadow_ray, shadow_prd);

			if (!shadow_prd.inShadow)
			{
				const float A = length(cross(light.v1, light.v2));
				// convert area based pdf to solid angle
				const float weight = nDl * LnDl * A / (M_PIf * Ldist * Ldist);
				result += light.emission * weight;
			}
		}
	}

	current_prd.radiance = result;
}

//-----------------------------------------------------------------------------
//
//  Shadow any-hit
//
//-----------------------------------------------------------------------------

rtDeclareVariable(PerRayData_pathtrace_shadow, current_prd_shadow, rtPayload, );

RT_PROGRAM void shadow()
{
	const float3 alpha_tex_sample = make_float3(tex2D(D_map, texcoord.x, texcoord.y));
	if (alpha_tex_sample.x == 0.0f)
	{
		rtIgnoreIntersection();
	}
	else
	{
		current_prd_shadow.inShadow = true;
		rtTerminateRay();
	}
}

RT_PROGRAM void any_hit_radiance()
{
	const float3 alpha_tex_sample = make_float3(tex2D(D_map, texcoord.x, texcoord.y));
	if (alpha_tex_sample.x == 0.0f)
	{
		rtIgnoreIntersection();
	}
}

//-----------------------------------------------------------------------------
//
//  Exception program
//
//-----------------------------------------------------------------------------

RT_PROGRAM void exception()
{
    output_buffer[launch_index] = make_float4(bad_color, 1.0f);
}


//-----------------------------------------------------------------------------
//
//  Miss program
//
//-----------------------------------------------------------------------------

RT_PROGRAM void miss()
{
	current_prd.radiance = bg_color;
    current_prd.done = true;
	current_prd.missed = true;
}


