#include "hip/hip_runtime.h"
/*
* Copyright (c) 2017 NVIDIA CORPORATION. All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions
* are met:
*  * Redistributions of source code must retain the above copyright
*    notice, this list of conditions and the following disclaimer.
*  * Redistributions in binary form must reproduce the above copyright
*    notice, this list of conditions and the following disclaimer in the
*    documentation and/or other materials provided with the distribution.
*  * Neither the name of NVIDIA CORPORATION nor the names of its
*    contributors may be used to endorse or promote products derived
*    from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
* PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
* CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
* EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
* PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
* OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <optixu/optixu_math_namespace.h>
#include "optixPathTracer.h"
#include "random.h"

//#include "VarianceAdaptive.h"
#include "HoelderAdaptive.h"
#include "MitchellFilterDevice.h"

using namespace optix;

struct PerRayData_pathtrace
{
	float3 result;
	float3 radiance;
	float3 attenuation;
	float3 origin;
	float3 direction;
	unsigned int seed;
	int depth;
	int countEmitted;
	int done;
	int missed;
	//int isAdaptive;
};

struct PerRayData_pathtrace_shadow
{
	bool inShadow;
};

// Scene wide variables
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(float, far_plane, , );
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

rtDeclareVariable(PerRayData_pathtrace, current_prd, rtPayload, );



//-----------------------------------------------------------------------------
//
//  Camera program -- main ray tracing loop
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtDeclareVariable(unsigned int, frame_number, , );
rtDeclareVariable(unsigned int, num_samples, , );
rtDeclareVariable(unsigned int, sqrt_num_samples, , );
rtDeclareVariable(unsigned int, rr_begin_depth, , );
rtDeclareVariable(unsigned int, pathtrace_ray_type, , );

RT_PROGRAM void pathtrace_camera_adaptive()
{

	// Debug!
	depth_gradient_buffer[launch_index] = make_float4(0.0f);
	size_t2 screen = output_buffer.size();

	float2 inv_screen = 1.0f / make_float2(screen) * 2.f;
	float2 pixel = (make_float2(launch_index)) * inv_screen - 1.f;

	float2 jitter_scale = inv_screen / sqrt_num_samples;

	initializeHoelderAdaptiveSceneDepthBuffer(launch_index, frame_number, camera_changed);
	initialize_hoelder_adaptive_buffers(launch_index, frame_number, camera_changed, static_cast<uint>(window_size));

	unsigned int adaptive_samples_per_pixel = compute_current_samples_number(launch_index, window_size_buffer[launch_index].x);
	//unsigned int adaptive_samples_per_pixel = 0;
	unsigned int current_samples_per_pixel = adaptive_samples_per_pixel;
	float3 result = make_float3(0.0f);

	unsigned int adaptive_sqrt_num_samples = sqrtf(static_cast<float>(adaptive_samples_per_pixel));

	if (!adaptive_sqrt_num_samples)
	{
		++adaptive_sqrt_num_samples;
	}

	unsigned int seed = tea<16>(screen.x*launch_index.y + launch_index.x, frame_number);

	int num_not_missed_rays = current_samples_per_pixel;

	if (current_samples_per_pixel)
	{
		//post_process_output_buffer[launch_index] = hoelder_adaptive_buffers[input_scene_render_buffer][launch_index];
		do
		{
			//
			// Sample pixel using jittering
			//
			unsigned int x = adaptive_samples_per_pixel % adaptive_sqrt_num_samples;
			unsigned int y = adaptive_samples_per_pixel / adaptive_sqrt_num_samples;
			float2 jitter = make_float2(x - rnd(seed), y - rnd(seed));
			float2 d = pixel + jitter*jitter_scale;
			float3 ray_origin = eye;
			float3 ray_direction = normalize(d.x*U + d.y*V + W);

			//rtPrintf("2D ray direction in screen coordinates: [ %f , %f ]\n", d.x, d.y);

			// Initialze per-ray data
			PerRayData_pathtrace prd;
			prd.result = make_float3(0.f);
			prd.attenuation = make_float3(1.f);
			prd.countEmitted = true;
			prd.done = false;
			prd.seed = seed;
			prd.depth = 0;
			//prd.isAdaptive = 1;
			prd.missed = 0;

			// Each iteration is a segment of the ray path.  The closest hit will
			// return new segments to be traced here.
			for (;;)
			{
				if (prd.depth == 1)
				{
					float ray_length = fabsf(length((prd.origin - eye)));
					float normalized_ray_length = ray_length / far_plane;//2500.0f;

					float a = 1.0f / (float)output_current_total_rays_buffer[launch_index].x;
					float3 old_depth = make_float3(output_scene_depth_buffer[launch_index]);
					output_scene_depth_buffer[launch_index] = make_float4(make_float3(min(normalized_ray_length, old_depth.x)), 1.0f);// make_float4(lerp(old_depth, make_float3(normalized_ray_length), a), 1.0f);

					hoelder_adaptive_scene_depth_buffer[launch_index] = make_float4(make_float3(min(normalized_ray_length, old_depth.x)), 1.0f);//make_float4(lerp(old_depth, make_float3(normalized_ray_length), a), 1.0f);

				}

				Ray ray = make_Ray(ray_origin, ray_direction, pathtrace_ray_type, scene_epsilon, RT_DEFAULT_MAX);
				rtTrace(top_object, ray, prd);

				if (prd.done)
				{
					if (prd.missed && prd.depth == 0)
					{
						num_not_missed_rays--;
					}
					// We have hit the background or a luminaire
					prd.result += prd.radiance * prd.attenuation;
					break;
				}

				// Russian roulette termination 
				if (prd.depth >= rr_begin_depth)
				{
					float pcont = fmaxf(prd.attenuation);
					if (rnd(prd.seed) >= pcont)
						break;
					prd.attenuation /= pcont;
				}

				prd.depth++;
				prd.result += prd.radiance * prd.attenuation;

				// Update ray data for the next path segment
				ray_origin = prd.origin;
				ray_direction = prd.direction;
			}

			float2 sample = make_float2(launch_index.x + jitter.x, launch_index.y + jitter.y);

			seed = prd.seed;
			output_current_total_rays_buffer[launch_index].x++;
			int current_total_rays = output_current_total_rays_buffer[launch_index].x;
			computeMitchellFilterSampleContributionInNeighborhood(sample, launch_index, prd.result, screen, current_total_rays, &output_filter_sum_buffer, &output_filter_x_sample_sum_buffer);
		} while (--current_samples_per_pixel);
	}
	//
	// Update the output buffer
	//

	if (num_not_missed_rays > 0)
	{
		evaluatePixelFileringEquation(launch_index, &output_buffer, &output_filter_sum_buffer, &output_filter_x_sample_sum_buffer);
	}
}

//
// Adaptive version of pathtracing end
//
