#include "hip/hip_runtime.h"
/*
* Copyright (c) 2017 NVIDIA CORPORATION. All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions
* are met:
*  * Redistributions of source code must retain the above copyright
*    notice, this list of conditions and the following disclaimer.
*  * Redistributions in binary form must reproduce the above copyright
*    notice, this list of conditions and the following disclaimer in the
*    documentation and/or other materials provided with the distribution.
*  * Neither the name of NVIDIA CORPORATION nor the names of its
*    contributors may be used to endorse or promote products derived
*    from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
* PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
* CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
* EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
* PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
* OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <optixu/optixu_math_namespace.h>
#include "optixPathTracer.h"
#include "random.h"
#include "VarianceAdaptive.h"

using namespace optix;

struct PerRayData_pathtrace
{
	float3 result;
	float3 radiance;
	float3 attenuation;
	float3 origin;
	float3 direction;
	unsigned int seed;
	int depth;
	int countEmitted;
	int done;
	//int isAdaptive;
};

struct PerRayData_pathtrace_shadow
{
	bool inShadow;
};

// Scene wide variables
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(float, far_plane, , );
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

rtDeclareVariable(PerRayData_pathtrace, current_prd, rtPayload, );



//-----------------------------------------------------------------------------
//
//  Camera program -- main ray tracing loop
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtDeclareVariable(unsigned int, frame_number, , );
rtDeclareVariable(unsigned int, sqrt_num_samples, , );
rtDeclareVariable(unsigned int, rr_begin_depth, , );
rtDeclareVariable(unsigned int, pathtrace_ray_type, , );

// Adaptive post processing variables and buffers

//rtDeclareVariable(unsigned int, window_size, , );
//rtDeclareVariable(unsigned int, max_ray_budget_total, , ) = static_cast<uint>(50u);
//rtDeclareVariable(unsigned int, max_per_frame_samples_budget, , ) = static_cast<uint>(5u);		/* this variable can be written by the user */
//rtDeclareVariable(int, camera_changed, , );

//
// Adaptive version of pathtracing begin
//

//rtDeclareVariable(VarianceAdaptive, variance_adaptive, , );// = VarianceAdaptive();

/*--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------*/
/* Adaptive additional rays variables */
//rtDeclareVariable(unsigned int, max_per_frame_samples_budget, , ) = static_cast<uint>(5u);		/* this variable will be written by the user */
//rtBuffer<int4, 2>	  adaptive_samples_budget_buffer;									/* this buffer will be initialized by the host, but must also be modified by the graphics device */
//rtBuffer<int4, 2>	  hoelder_refinement_buffer;										/* this buffer contains the information, where refinement will take place according to
																						//hoelder regularity criterion, everywhere where refinement is needed value is 1, else zero */

rtBuffer<float4, 2>	  hoelder_refinement_buffer;

rtBuffer<int4, 2>	  window_size_buffer;

//rtBuffer<float4, 2>	  per_window_variance_buffer_output;

//rtBuffer<float4, 2>   input_buffer;														/* this buffer contains the initially rendered picture to be post processed */
rtBuffer<float4, 2>   input_scene_depth_buffer;											/* this buffer contains the necessary depth values to compute the gradient 
																						via finite differences for the hoelder alpha computation via the smooth regime */
rtBuffer<float4, 2>   hoelder_adaptive_scene_depth_buffer;								/* this buffer contains only the depth values of the adaptive samples which has been evaluated 
																						and is used for gradient computation */
rtBuffer<float4, 2>   post_process_output_buffer;										/* this buffer contains the result, processed with additional adaptive rays */

// For debug!
rtBuffer<float4, 2>   depth_gradient_buffer;
// For debug!
rtBuffer<float4, 2>   hoelder_alpha_buffer;
// For debug!
rtBuffer<float4, 2>   total_sample_count_buffer;



//
// H�dler Adaptive Image Synthesis (begin)
//

// non-smooth regime
//static __device__ __inline__ float compute_window_hoelder_non_smooth_regime(uint2 center, uint window_size)
//{
//	size_t2 screen = input_buffer.size();
//
//	float alpha = 100.f;
//
//	uint squared_window_size = window_size * window_size;
//	uint half_window_size = (window_size / 2) + (window_size % 2);
//	uint2 top_left_window_corner = make_uint2(center.x - half_window_size, center.y - half_window_size);
//
//	//rtPrintf("\nTop left window corner: [ %d, %d ]\n", top_left_window_corner.x, top_left_window_corner.y);
//
//	float3 center_buffer_val = make_float3(input_buffer[center].x, input_buffer[center].y, input_buffer[center].z);
//	float centerColorMean = 1.f / 3.f * (center_buffer_val.x + center_buffer_val.y + center_buffer_val.z);
//	float neighborColorMean = 0.0f;
//
//	/* compute mean value */
//	for (uint i = 0; i < squared_window_size; i++)
//	{
//		uint2 idx = make_uint2((i % window_size + top_left_window_corner.x) % screen.x, (i / window_size + top_left_window_corner.y) % screen.y);
//		if (idx.x == center.x && idx.y == center.y)
//		{
//			continue;
//		}
//		float3 neighbor_buffer_val = make_float3(input_buffer[idx].x, input_buffer[idx].y, input_buffer[idx].z);
//		neighborColorMean = 1.f / 3.f * (neighbor_buffer_val.x + neighbor_buffer_val.y + neighbor_buffer_val.z);
//
//		float neighbor_center_distance = length(make_float2(static_cast<float>(center.x) - static_cast<float>(idx.x), static_cast<float>(center.y) - static_cast<float>(idx.y)));
//
//		float log_base = log(fabs(neighbor_center_distance) + 1.0f);
//
//		if (log_base != 0.0f)
//		{
//			float log_x = log(fabs(1.0f / 2.0f /*hoelder constant, also try value 3*/ * (centerColorMean - neighborColorMean)) + 1.0f);
//			alpha = min(alpha, (log_x / log_base));
//			alpha = clamp(alpha, 0.0f, 100.f);
//		}
//	}
//	//rtPrintf("___________________________________________________________________________________________\n\n\n");
//
//	return alpha;
//};
//
//// modulo border treatment
//static __device__ __inline__ float3 compute_color_gradient(uint2 idx)
//{
//	size_t2 screen = input_scene_depth_buffer.size();
//
//	uint2 idx_up = make_uint2(idx.x, idx.y + 1 % screen.y);
//	uint2 idx_down = make_uint2(idx.x, min(0, idx.y - 1));//idx.y - 1 < 0 ? screen.y : idx.y - 1);
//
//	//uint2 idx_left = make_uint2(idx.x - 1 < 0 ? screen.x : idx.x - 1, idx.y);
//	uint2 idx_left = make_uint2(min(0, idx.x - 1), idx.y);
//	uint2 idx_right = make_uint2(idx.x + 1 % screen.x, idx.y);
//
//	float4 gradient_y = input_scene_depth_buffer[idx_up] - input_scene_depth_buffer[idx_down];
//	float4 gradient_x = input_scene_depth_buffer[idx_right] - input_scene_depth_buffer[idx_left];
//
//	float4 gradient_tmp = gradient_y + gradient_x;
//
//	float3 gradient = make_float3(gradient_tmp.x / 2.0f, gradient_tmp.y / 2.0f, gradient_tmp.z / 2.0f);
//
//	return gradient;
//};
//
// modulo border treatment
static __device__ __inline__ float3 compute_depth_gradient(uint2 idx)
{
	size_t2 screen = input_buffer.size();

	uint2 idx_up = make_uint2(idx.x, idx.y + 1 % screen.y);
	uint2 idx_down = make_uint2(idx.x, min(0, idx.y - 1));
	uint2 idx_left = make_uint2(min(0, idx.x - 1), idx.y);
	uint2 idx_right = make_uint2(idx.x + 1 % screen.x, idx.y);

	float4 gradient_y = input_buffer[idx_up] - input_buffer[idx_down];
	float4 gradient_x = input_buffer[idx_right] - input_buffer[idx_left];

	float4 gradient_tmp = gradient_y + gradient_x;

	float3 gradient = make_float3(gradient_tmp.x / 2.0f, gradient_tmp.y / 2.0f, gradient_tmp.z / 2.0f);

	return gradient;
};

// modulo border treatment
// first three values of float4 return are the color gradient
// last value of float4 return is the depth/geometry gradient
static __device__ __inline__ float4 compute_color_depth_gradient(uint2 idx)
{
	uint2 screen = make_uint2(input_buffer.size().x, input_buffer.size().y);

	int up = min(idx.y + 1, screen.y);
	int down = max(0, static_cast<int>(idx.y) - 1);
	int left = max(0, static_cast<int>(idx.x) - 1);
	int right = min(idx.x + 1, screen.x);
/*
	if (up > screen.y)
	{
		printf("Up is bigger than screen.y!: %d\n\n", up);
	}
	if (down < 0)
	{
		printf("Down is smaller than 0!: %d\n\n", down);
	}
	if (left < 0)
	{
		printf("Left is smaller than 0!: %d\n\n", left);
	}
	if (right > screen.x)
	{
		printf("Right is bigger than screen.x!: %d\n\n", right);
	}*/

	uint2 idx_up = make_uint2(idx.x, static_cast<uint>(up));
	uint2 idx_down = make_uint2(idx.x, static_cast<uint>(down));
	uint2 idx_left = make_uint2(static_cast<uint>(left), idx.y);
	uint2 idx_right = make_uint2(static_cast<uint>(right), idx.y);

	float4 gradient_color_y = input_buffer[idx_up] - input_buffer[idx_down];
	float4 gradient_color_x = input_buffer[idx_right] - input_buffer[idx_left];

	float4 gradient_color_tmp = gradient_color_y + gradient_color_x;

	float3 gradient_color = make_float3(0.5f * gradient_color_tmp.x, 0.5f * gradient_color_tmp.y, 0.5f * gradient_color_tmp.z);

	float gradient_depth_x = input_scene_depth_buffer[idx_up].x - input_scene_depth_buffer[idx_down].x;
	float gradient_depth_y = input_scene_depth_buffer[idx_right].x - input_scene_depth_buffer[idx_left].x;

	//float gradient_depth_x = hoelder_adaptive_scene_depth_buffer[idx_up].x - hoelder_adaptive_scene_depth_buffer[idx_down].x;
	//float gradient_depth_y = hoelder_adaptive_scene_depth_buffer[idx_right].x - hoelder_adaptive_scene_depth_buffer[idx_left].x;

	float gradient_depth = gradient_depth_x + gradient_depth_y;

	float4 combined_gradient = make_float4(gradient_color.x, gradient_color.y, gradient_color.z, gradient_depth);

	return combined_gradient;
};

static __device__ __inline__ float compute_window_hoelder_smooth_regime(uint2 center, uint window_size)
{
	size_t2 screen = input_buffer.size();

	float alpha = 100.f;

	uint squared_window_size = window_size * window_size;
	uint half_window_size = (window_size / 2) + (window_size % 2);
	uint2 top_left_window_corner = make_uint2(center.x - half_window_size, center.y - half_window_size);

	float3 center_buffer_val = make_float3(input_buffer[center].x, input_buffer[center].y, input_buffer[center].z);
	float centerColorMean = 1.f / 3.f * (center_buffer_val.x + center_buffer_val.y + center_buffer_val.z);
	float neighborColorMean = 0.0f;

	/* compute mean value */
	for (uint i = 0; i < squared_window_size; i++)
	{
		uint2 idx = make_uint2((i % window_size + top_left_window_corner.x) % screen.x, (i / window_size + top_left_window_corner.y) % screen.y);
		float3 neighbor_buffer_val = make_float3(input_buffer[idx].x, input_buffer[idx].y, input_buffer[idx].z);
		neighborColorMean = 1.f / 3.f * (neighbor_buffer_val.x + neighbor_buffer_val.y + neighbor_buffer_val.z);

		/*float gradient_of_mean_color = length(compute_color_gradient(idx));*/
		float gradient_of_mean_color = length(compute_depth_gradient(idx));

		float neighbor_center_distance = length(make_float2(static_cast<float>(center.x) - static_cast<float>(idx.x), static_cast<float>(center.y) - static_cast<float>(idx.y)));

		float log_base = log(fabs(neighbor_center_distance) + 1.0f);

		if (log_base != 0.0f)
		{
			float log_x = log(fabs(1.0f / 2.0f /*hoelder constant, also try value 3*/ * (centerColorMean - neighborColorMean - gradient_of_mean_color * neighbor_center_distance) + 1.0f));

			alpha = min(alpha, log_x / log_base);

			alpha = clamp(alpha, 0.0f, 100.f);
		}
	}

	return alpha;
};

static __device__ __inline__ float compute_window_hoelder(uint2 center, uint window_size)
{
	size_t2 screen = input_buffer.size();

	float alpha = 100.f;

	uint squared_window_size = window_size * window_size;
	uint half_window_size = (window_size / 2) + (window_size % 2);
	uint2 top_left_window_corner = make_uint2(center.x - half_window_size, center.y - half_window_size);

	float3 center_buffer_val = make_float3(input_buffer[center].x, input_buffer[center].y, input_buffer[center].z);
	float centerColorMean = 1.f / 3.f * (center_buffer_val.x + center_buffer_val.y + center_buffer_val.z);
	float neighborColorMean = 0.0f;

	for (uint i = 0; i < squared_window_size; i++)
	{
		uint2 idx = make_uint2((i % window_size + top_left_window_corner.x) % screen.x, (i / window_size + top_left_window_corner.y) % screen.y);

		float4 color_depth_gradient = compute_color_depth_gradient(idx);

		// Debug!
		depth_gradient_buffer[idx] = make_float4(color_depth_gradient.w);

		float neighbor_center_distance = length(make_float2(static_cast<float>(center.x) - static_cast<float>(idx.x), static_cast<float>(center.y) - static_cast<float>(idx.y)));

		float log_base = log(fabsf(neighbor_center_distance) + 1.0f);

		//if (i % window_size <= i / window_size)
		//{
		//	float inverseWindowSize = 1.0f / static_cast<float>(window_size);
		//	post_process_output_buffer[idx] = make_float4(window_size * inverseWindowSize, 0.0f, window_size_buffer[idx].x * inverseWindowSize, 1.0f);
		//}

		//if (log_base == 0.0f)
		//{
		//	rtPrintf("Neighbor center distance: || [ %f , %f ]-[ %f , %f ] || = %f \n\n", 
		//		static_cast<float>(center.x), static_cast<float>(center.y), 
		//		static_cast<float>(idx.x), static_cast<float>(idx.y), 
		//		neighbor_center_distance);
		//}

		if (log_base != 0.0f)
		{
			float3 neighbor_buffer_val = make_float3(input_buffer[idx].x, input_buffer[idx].y, input_buffer[idx].z);
			neighborColorMean = 1.f / 3.f * (neighbor_buffer_val.x + neighbor_buffer_val.y + neighbor_buffer_val.z);
			//float log_x = log(fabs(1.0f / 2.0f /*hoelder constant, also try value 3*/ * (centerColorMean - neighborColorMean - gradient_of_mean_color * neighbor_center_distance) + 1.0f));
			float log_x = 0.0f;

			// Decide whether to use smooth or non-smooth regime based on depth/geometry buffer map. 
			// Where there is a very small depth/geometry gradient use smooth regime computation hoelder alpha, 
			// else use non-smooth regime hoelder alpha computation (log_x value makes for that distinction). 
			if (fabsf(color_depth_gradient.w)/* Value 'w' is depth/geometry gradient */ <= 0.01f/* Currently more or less arbitary threshhold for an edge! */)
			{
				//post_process_output_buffer[idx] = make_float4(100.0f, 0.0f, 100.0f, 1.0f);
				//rtPrintf("\nsmooth!!!\n");
				float3 color_gradient = make_float3(color_depth_gradient.x, color_depth_gradient.y, color_depth_gradient.z);
				float mean_of_color_gradient = length(color_gradient);
				log_x = log(fabsf(1.0f / 2.0f /*hoelder constant, also try value 3*/ * (centerColorMean - neighborColorMean - mean_of_color_gradient * neighbor_center_distance)) + 1.0f);
			}
			else
			{
				//post_process_output_buffer[idx] = make_float4(100.0f, 0.0f, 100.0f, 1.0f);
				//rtPrintf("\nnon-smooth!!!\n");
				float log_x = log(fabsf(1.0f / 2.0f /*hoelder constant, also try value 3*/ * (centerColorMean - neighborColorMean)) + 1.0f);
			}

			//if (idx.x == 0)
			//{
			//	rtPrintf("Temporary alpha: %f\n\n", (log_x / log_base));
			//}
			//if (idx.x == 1 && idx.y == 1)
			//{
			//	rtPrintf("\n\n");
			//}
			

			alpha = min(alpha, log_x / log_base);
			alpha = clamp(alpha, 0.0f, 100.f);
		}
	}

	return alpha;
};

static __device__ __inline__ float4 hoelder_refinement(float alpha, uint2 center, uint window_size)
{
	float4 alphas = make_float4(0.0f);

	uint2 center1 = center + make_uint2((center.x - 0.5 * center.x), (center.y - 0.5 * center.y));
	uint2 center2 = center + make_uint2((center.x + 0.5 * center.x), (center.y - 0.5 * center.y));
	uint2 center3 = center + make_uint2((center.x - 0.5 * center.x), (center.y + 0.5 * center.y));
	uint2 center4 = center + make_uint2((center.x + 0.5 * center.x), (center.y + 0.5 * center.y));

	uint half_window_size = 0.5f * window_size;

	if (alpha < 0.5f /* Arbitary alpha threshold */)
	{
		alphas.x = compute_window_hoelder(center1, half_window_size);
		alphas.y = compute_window_hoelder(center2, half_window_size);
		alphas.z = compute_window_hoelder(center3, half_window_size);
		alphas.w = compute_window_hoelder(center4, half_window_size);
	}

	return alphas;
}

// TODO: Rename this function to something more general, like "expend_samples_of_sample_map"
static __device__ __inline__ uint compute_hoelder_samples_number(uint2 current_launch_index, uint window_size)
{
	//rtPrintf("Hoelder alpha: %f\n\n", alpha);

	//float oversampling_factor = 1.25f;

	uint samples_number = min(adaptive_samples_budget_buffer[current_launch_index].x, max_per_frame_samples_budget);

	//rtPrintf("Currently avaible adaptive samples: %d\n\n", samples_number);

	if (adaptive_samples_budget_buffer[current_launch_index].x > 0)
	{
		//samples_number = static_cast<uint>(clamp(static_cast<float>(samples_number), 0.0f, static_cast<float>(max_per_frame_samples_budget)));
		adaptive_samples_budget_buffer[current_launch_index] = make_int4(adaptive_samples_budget_buffer[current_launch_index].x - static_cast<int>(samples_number));
	}

	//rtPrintf("Currently avaible adaptive samples: %d\n\n", adaptive_samples_budget_buffer[current_launch_index].x);

	return samples_number;
};

static __device__ __inline__ uint hoelder_compute_current_samples_number_and_manage_buffers(uint2 current_launch_index, uint2 current_window_center, uint window_size)
{
	if (window_size >= 2)
	{
		float hoelder_alpha = -1.0f;
		float hoelder_alpha_no_refinement_threshhold = 0.5f;

		//rtPrintf("Hoelder refinement buffer value: %d\n", hoelder_refinement_buffer[current_launch_index].x);
		//if (adaptive_samples_budget_buffer[current_launch_index].x == 1)
		//{
		//	rtPrintf("Currently avaible adaptive samples: %d\n\n", adaptive_samples_budget_buffer[current_launch_index].x);
		//}
		/*rtPrintf("Currently avaible adaptive samples: %d\n\n", adaptive_samples_budget_buffer[current_launch_index].x);*/

		if (hoelder_refinement_buffer[current_launch_index].x == 1)
		{
			//rtPrintf("Compute!!!\n\n");
			hoelder_alpha = compute_window_hoelder(current_window_center, window_size);
			//hoelder_refinement_buffer[current_launch_index] = make_int4(0);
			hoelder_refinement_buffer[current_launch_index] = make_float4(0);

			//printf("Current hoelder: %f\n\n", hoelder_alpha);
		}

		//rtPrintf("Current hoelder: %f\n\n", hoelder_alpha);

		hoelder_alpha_buffer[current_launch_index] = make_float4(hoelder_alpha * 10.0f);

		if (hoelder_alpha < 0.0f)
		{
			//rtPrintf("Set to one hundred!!!\n\n");
			hoelder_alpha = 100.0f;
		}

		if (hoelder_alpha * 100.0f < hoelder_alpha_no_refinement_threshhold)
		{
			//printf("Current hoelder: %f\n\n", hoelder_alpha);
			//rtPrintf("Refine next frame!!!\n\n");
			//hoelder_refinement_buffer[current_launch_index] = make_int4(1);
			hoelder_refinement_buffer[current_launch_index] = make_float4(1);
			adaptive_samples_budget_buffer[current_launch_index] += make_int4(1);// hoelder_refinement_buffer[current_launch_index];
			//total_sample_count_buffer[current_launch_index] += make_float4(1.0f/log2f(static_cast<float>(window_size) + 1));
			window_size_buffer[current_launch_index] = make_int4(0.5f * window_size_buffer[current_launch_index].x);
			//rtPrintf("Currently avaible adaptive samples: %d\n\n", adaptive_samples_budget_buffer[current_launch_index].x);
		}
	}

	return compute_hoelder_samples_number(current_launch_index, window_size);
	//return 0;
};

static __device__ __inline__ void initialize_hoelder_refinement_buffer(uint2 current_launch_index, int frame_number, int camera_changed, uint window_size)
{
	if (frame_number == 1 || camera_changed == 1)
	{
		//rtPrintf("Initialize holder refinement buffer!!!\n\n");
		//hoelder_refinement_buffer[current_launch_index] = make_int4(1);
		hoelder_refinement_buffer[current_launch_index] = make_float4(1);
		adaptive_samples_budget_buffer[current_launch_index] = make_int4(0);
		total_sample_count_buffer[current_launch_index] = make_float4(0.0f);

		window_size_buffer[current_launch_index] = make_int4(window_size);
		//rtPrintf("Currently avaible adaptive samples: %d\n\n", adaptive_samples_budget_buffer[current_launch_index].x);
	}
};

static __device__ __inline__ void initializeHoelderAdaptiveSceneDepthBuffer(uint2 current_launch_index, int frame_number, int camera_changed)
{
	if (frame_number == 1 || camera_changed == 1)
	{
		//rtPrintf("Init!!!\n\n");
		hoelder_adaptive_scene_depth_buffer[current_launch_index] = input_scene_depth_buffer[current_launch_index];
	}
};

static __device__ __inline__ void resetHoelderAdaptiveSceneDepthBuffer(uint2 current_launch_index)
{
	hoelder_adaptive_scene_depth_buffer[current_launch_index] = make_float4(0.0f);
};

//
// H�dler Adaptive Image Synthesis (end)
//

static __device__ __inline__ uint compute_current_samples_number(uint2 current_launch_index, uint window_size)
{
	uint sample_number = 0;

	//uint additional_samples_number = 0;

	size_t2 screen = input_buffer.size();

	uint times_width = screen.x / window_size;
	uint times_height = screen.y / window_size;

	uint horizontal_padding = static_cast<uint>(0.5f * (screen.x - (times_width * window_size)));
	uint vertical_padding = static_cast<uint>(0.5f * (screen.y - (times_height * window_size)));

	uint half_window_size = (window_size / 2) + (window_size % 2);

	uint2 times_launch_index = make_uint2(((current_launch_index.x / window_size) * window_size) % screen.x, ((current_launch_index.y / window_size) * window_size) % screen.y);

	uint2 current_window_center = make_uint2(times_launch_index.x + horizontal_padding + half_window_size, times_launch_index.y + vertical_padding + half_window_size);

	//float variance = compute_window_variance(current_window_center, window_size);

	//float hoelder_alpha = compute_window_hoelder(current_window_center, window_size);

	//hoelder_alpha_buffer[current_launch_index] = make_float4(hoelder_alpha * 100.0f);

	//sample_number = compute_samples_number(current_launch_index, (30.0f * variance));

	//sample_number = compute_hoelder_samples_number(current_launch_index, (10.0f * hoelder_alpha), window_size);

	sample_number = hoelder_compute_current_samples_number_and_manage_buffers(current_launch_index, current_window_center, window_size);
	float sample_count_fraction = static_cast<float>(sample_number) / log2f(static_cast<float>(window_size) + 50);
	total_sample_count_buffer[current_launch_index] += make_float4(sample_count_fraction, 0.0f, 0.0f, 1.0f);

	//hoelder_refinement(hoelder_alpha, current_window_center, window_size);

	rtPrintf("Sample number: %d\n\n", sample_number);

	return sample_number;
};

//static __device__ __inline__ void init_variance_adaptive_struct(VarianceAdaptiveStruct* variance_adaptive_struct)
//{
//	variance_adaptive_struct->_adaptive_samples_budget_buffer = &adaptive_samples_budget_buffer;
//	variance_adaptive_struct->_input_buffer = &input_buffer;
//	variance_adaptive_struct->_per_window_variance_buffer_output = &per_window_variance_buffer_output;
//
//	variance_adaptive_struct->_max_per_frame_samples_budget = max_per_frame_samples_budget;
//	variance_adaptive_struct->_window_size = window_size;
//};

RT_PROGRAM void pathtrace_camera_adaptive()
{
	//rtPrintf("Current samples number: %d\n\n", adaptive_samples_budget_buffer[launch_index].x);

	// Debug!
	depth_gradient_buffer[launch_index] = make_float4(0.0f);

	size_t2 screen = input_buffer.size();

	float2 inv_screen = 1.0f / make_float2(screen) * 2.f;
	float2 pixel = (make_float2(launch_index)) * inv_screen - 1.f;

	float2 jitter_scale = inv_screen / sqrt_num_samples;

	//initializeHoelderAdaptiveSceneDepthBuffer(launch_index, frame_number, camera_changed);
	//initialize_hoelder_refinement_buffer(launch_index, frame_number, camera_changed, static_cast<uint>(window_size));

	//VarianceAdaptiveStruct variance_adaptive_struct;
	//init_variance_adaptive_struct(&variance_adaptive_struct);

	//unsigned int adaptive_samples_per_pixel = compute_current_samples_number(launch_index, variance_adaptive_struct);//, window_size, max_per_frame_samples_budget);//compute_current_samples_number(launch_index, window_size_buffer[launch_index].x, 
																			//&input_buffer, 
																			//&adaptive_samples_budget_buffer, max_per_frame_samples_budget, &per_window_variance_buffer_output);
	unsigned int adaptive_samples_per_pixel = compute_current_samples_number(launch_index, window_size);
	unsigned int current_samples_per_pixel = adaptive_samples_per_pixel;
	float3 result = make_float3(0.0f);

	unsigned int adaptive_sqrt_num_samples = sqrtf(static_cast<float>(adaptive_samples_per_pixel));

	if (!adaptive_sqrt_num_samples)
	{
		++adaptive_sqrt_num_samples;
	}

	unsigned int seed = tea<16>(screen.x*launch_index.y + launch_index.x, frame_number);

	float3 pixel_color = make_float3(input_buffer[launch_index]);

	//resetHoelderAdaptiveSceneDepthBuffer(launch_index);

	if (current_samples_per_pixel)
	{
		do
		{
			//
			// Sample pixel using jittering
			//
			unsigned int x = adaptive_samples_per_pixel % adaptive_sqrt_num_samples;
			unsigned int y = adaptive_samples_per_pixel / adaptive_sqrt_num_samples;
			float2 jitter = make_float2(x - rnd(seed), y - rnd(seed));
			float2 d = pixel + jitter*jitter_scale;
			float3 ray_origin = eye;
			float3 ray_direction = normalize(d.x*U + d.y*V + W);

			// Initialze per-ray data
			PerRayData_pathtrace prd;
			prd.result = make_float3(0.f);
			prd.attenuation = make_float3(1.f);
			prd.countEmitted = true;
			prd.done = false;
			prd.seed = seed;
			prd.depth = 0;
			//prd.isAdaptive = 1;

			// Each iteration is a segment of the ray path.  The closest hit will
			// return new segments to be traced here.
			for (;;)
			{
				if (prd.depth == 1)
				{
					float ray_length = fabsf(length((prd.origin - eye)));
					float normalized_ray_length = ray_length / far_plane;//2500.0f;

					float a = 1.0f / (float)frame_number;
					float3 old_depth = make_float3(input_scene_depth_buffer[launch_index]);
					input_scene_depth_buffer[launch_index] = make_float4(lerp(old_depth, make_float3(normalized_ray_length), a), 1.0f);

					hoelder_adaptive_scene_depth_buffer[launch_index] = make_float4(make_float3(normalized_ray_length), 1.0f);
					//if (frame_number == 1)
					//{
					//	input_scene_depth_buffer[launch_index] = make_float4(normalized_ray_length);
					//}
				}

				//if (prd.depth == 1)
				//{
				//	float ray_length = fabsf(length((prd.origin - eye)));
				//	float normalized_ray_length = ray_length / far_plane;//2500.0f;

				//	hoelder_adaptive_scene_depth_buffer[launch_index] = make_float4(make_float3(normalized_ray_length), 1.0f);
				//}

				Ray ray = make_Ray(ray_origin, ray_direction, pathtrace_ray_type, scene_epsilon, RT_DEFAULT_MAX);
				rtTrace(top_object, ray, prd);

				if (prd.done)
				{
					// We have hit the background or a luminaire
					prd.result += prd.radiance * prd.attenuation;
					break;
				}

				// Russian roulette termination 
				if (prd.depth >= rr_begin_depth)
				{
					float pcont = fmaxf(prd.attenuation);
					if (rnd(prd.seed) >= pcont)
						break;
					prd.attenuation /= pcont;
				}

				prd.depth++;
				prd.result += prd.radiance * prd.attenuation;

				// Update ray data for the next path segment
				ray_origin = prd.origin;
				ray_direction = prd.direction;
			}

			result += prd.result;
			seed = prd.seed;
		} while (--current_samples_per_pixel);

		pixel_color = result / (adaptive_sqrt_num_samples*adaptive_sqrt_num_samples);

		// Pink coloring of tiles for debug
		//if (adaptive_samples_per_pixel == 1 && window_size_buffer[launch_index].x <= 4)
		//{
		//	pixel_color = make_float3(window_size, 0.0f, window_size_buffer[launch_index].x);
		//}

		//if (adaptive_samples_per_pixel >= 1)
		//{
		//	pixel_color = make_float3(0.0f);
		//}
	}
	//
	// Update the output buffer
	//

	float a = 1.0f / (float)frame_number;
	float3 old_color = make_float3(input_buffer[launch_index]);
	post_process_output_buffer[launch_index] = make_float4(lerp(old_color, pixel_color, a), 1.0f);

	//compute_current_window_test(launch_index, 5);
}

//
// Adaptive version of pathtracing end
//
