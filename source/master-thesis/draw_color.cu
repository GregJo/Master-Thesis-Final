#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2017 NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////   Adaptive Additional Rays Test   //////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

rtDeclareVariable(float3,		// type of the variable used in an RT_PROGRAM in this .cu file
	shading_normal,				// name of the variable used in an RT_PROGRAM in this .cu file
	attribute shading_normal,	// semantic variable declared on the API in the corresponding .cpp file, attribute is to specify that the variable is part of a struct
	);

// per ray data struct
struct PerRayData_radiance
{
	float3 result;				// struct variable carrying our calculated output
	float  importance;
	int depth;
	int done;
};

rtDeclareVariable(PerRayData_radiance,
	prd_radiance, 
	rtPayload,							//This is a semantic name, not an API declared variable name to bind user data to
	);

RT_PROGRAM void closest_hit_radiance0()
{
	prd_radiance.result = normalize(rtTransformNormal(	// transforms n as a normal using the current active transformation stack (the inverse transpose)
		RT_OBJECT_TO_WORLD,								// other option would be RT_WORLD_TO_OBJECT
		shading_normal))
		*0.5f + 0.5f;
	prd_radiance.done = true;
}

rtDeclareVariable(float3, bg_color, , );

// Miss program for a ray, in case a ray misses the geometry give it the background color.
RT_PROGRAM void miss()
{
	prd_radiance.result = bg_color;
	prd_radiance.done = true;
}

static __device__ __inline__ uchar4 make_color(const float3& c)
{
	return make_uchar4(static_cast<unsigned char>(__saturatef(c.z)*255.99f),  /* B */
	static_cast<unsigned char>(__saturatef(c.y)*255.99f),  /* G */
	static_cast<unsigned char>(__saturatef(c.x)*255.99f),  /* R */
	255u);                                                 /* A */
};

static __device__ __inline__ float3 revert_color(const uchar4& c)
{
	return make_float3(static_cast<unsigned char>(__saturatef(c.z)*1.0f/255.99f),  /* B */
		static_cast<unsigned char>(__saturatef(c.y)*1.0f/255.99f),  /* G */
		static_cast<unsigned char>(__saturatef(c.x)*1.0f/255.99f)  /* R */);                                                 /* A */
};

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );

rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtDeclareVariable(float3, bad_color, , );
rtBuffer<uchar4, 2>   output_buffer;

/*
For post processing create multiple ray genereation programs.
The first ray generation program provides the input image.
The other ray generation programs do custom work on the input data initially provided by the first  

Useful comment from NVIDIA guy on their dev talk forum:
"There always has always been an easy path to custom post-processing within optix -- your own ray-gen programs which do post-processing, as you mention, 
or your own CUDA kernels. 
The postprocessing allows you to add optix launches to the pipeline (either for rendering or custom postprocess operations) so that you can use the pipeline 
as your all-in-one per-frame render pipeline."
*/
RT_PROGRAM void pinhole_camera()
{
	size_t2 screen = output_buffer.size();

	float2 d = make_float2(launch_index) /
		make_float2(screen) * 2.f - 1.f;
	float3 ray_origin = eye;
	float3 ray_direction = normalize(d.x*U + d.y*V + W);

	Ray ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon);
	PerRayData_radiance prd;
	prd.importance = 1.f;
	prd.depth = 0;
	prd.done = false;

	rtTrace(top_object, ray, prd); /* find out when its done, its important to know whether the code proceeds after this line after "rtTrace" is 'finished',
								   or if it starts a parallel subroutine and the code advances without waiting for "rtTrace" to finish (i assume the latter,
								   due to what i read in the technical overview -> the former is true, evidence by testing) */

	output_buffer[launch_index] = make_color(prd.result);
}

///*--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------*/
///* Adaptive additional rays variables */
//rtDeclareVariable(uint, max_per_launch_idx_ray_budget, , ) = static_cast<uint>(5u);		/* this variable will be written by the user */
//rtBuffer<uchar4, 2>   additional_rays_buffer;											/* this buffer will be initialized by the host, but must also be modified by the graphics device */
//
//rtBuffer<uchar4, 2>   input_buffer;														/* this buffer contains the initially rendered picture to be post processed */
//rtBuffer<uchar4, 2>   post_process_output_buffer;										/* this buffer contains the result, processed with additional adaptive rays */
//
//rtDeclareVariable(float, window_size, , );
//
//static __device__ __inline__ float compute_window_variance(uint2 center, uint window_size)
//{
//	float mean = 0.f;
//	float variance = 0.f;
//	uint squared_window_size = window_size * window_size;
//	uint2 upper_top_left_window = make_uint2(center.x - static_cast<uint>(static_cast<float>(window_size) / 2.f), center.y - static_cast<uint>(static_cast<float>(window_size) / 2.f));
//	/* compute mean value */
//	for (uint i = 0; i < squared_window_size; i++)
//	{
//		uint2 idx = make_uint2(static_cast<uint>(i / window_size) + upper_top_left_window.x, static_cast<uint>(i % window_size) + upper_top_left_window.y);
//		float3 input_buffer_val = revert_color(input_buffer[idx]);
//		mean += 1.f/3.f * (input_buffer_val.x + input_buffer_val.y + input_buffer_val.z);
//	}
//
//	mean *= 1.f/ squared_window_size;
//
//	/* compute variance */
//	for (uint i = 0; i < squared_window_size; i++)
//	{
//		uint2 idx = make_uint2(static_cast<uint>(i / window_size) + upper_top_left_window.x, static_cast<uint>(i % window_size) + upper_top_left_window.y);
//		float3 input_buffer_val = revert_color(input_buffer[idx]);
//		float var = 1.f / 3.f * (input_buffer_val.x + input_buffer_val.y + input_buffer_val.z);
//		variance += var;
//	}
//
//	variance = 1.f / squared_window_size * (variance) - (mean * mean);
//
//	return variance;
//};
//
//static __device__ __inline__ uint compute_variance_based_additional_samples_number(uint window_size) 
//{
//	uint additional_samples_number = 0;
//	/* check if box window is in buffer window */
//	/* actually compute 'additional_samples_number' */
//	return additional_samples_number;
//};
//
//RT_PROGRAM void adaptive_camera()
//{
//	/* Testing for additional adaptive rays. Added jittering for test purposes. */
//	
//	/* 
//		Postpone launching additional rays until first currently traced ray output is avaible (extend to neighborhood after success).
//			- 1. Postponing will be done with a loop, which will run indefinitely and does nothing (maybe use observer pattern here, more elegant than having a loop with an if statement), 
//				 until a condition is met, in this case when the output buffer has been written (-> no longer necessary, because the code advances after "rtTrace" only after its done).
//			  2. Upon reaching the written output buffer state which i will modify the additional "additional_rays_buffer" values, which are initialized with ("max_per_launch_idx_ray_budget" + 1)
//			     so that they contain an arbitary smaller or value (but only corresponding (neighboring) values to the current launchIdx).
//			  3. After setting the current additional(, adaptive) ray budget i break/leave the loop and start another, that launches another loop, in which i launch additional rays,
//			     according to the current budget and add/write the results into the output buffer.
//		Additional adaptive rays count will be avaible in the "additional_rays_buffer"
//	*/
//	size_t2 screen = post_process_output_buffer.size();
//
//	float2 d = make_float2(launch_index) /
//		make_float2(screen) * 2.f - 1.f;
//
//	uint additional_rays_count = static_cast<uint>(additional_rays_buffer[launch_index].x);
//
//	float3 ray_origin = eye;
//	float3 ray_direction = normalize(d.x*U + d.y*V + W);
//
//	/* Make the following 'adaptive pass' test to a real adaptive pass (for that i must ensure, that the first resulting image is completely avaible). */
//	//if (prd.done)
//	//{
//		additional_rays_count = static_cast<uint>(input_buffer[launch_index].x) % (max_per_launch_idx_ray_budget + 1u);
//		//rtPrintf("Launch index: %u, %u; Additional rays count: %u !\n\n", launch_index.x, launch_index.y, additional_rays_count);
//		float jitter = static_cast<float>(additional_rays_count) / static_cast<float>(max_per_launch_idx_ray_budget);
//		float jitterScale = 0.1f;
//		jitter = jitter * jitterScale;
//
//		if (additional_rays_count <= 0)
//		{
//			post_process_output_buffer[launch_index] = make_color(bad_color);
//		}
//
//		while (additional_rays_count > 0u)
//		{
//			//rtPrintf("Additional rays left: %u !\n", additional_rays_count);
//			float3 jittered_ray_origin;
//
//			jittered_ray_origin.x = ray_origin.x + jitter;
//			jittered_ray_origin.y = ray_origin.y - jitter;
//			jittered_ray_origin.z = ray_origin.z + jitter;
//
//			float3 jittered_ray_direction;
//
//			jittered_ray_direction.x = ray_direction.x + jitter;
//			jittered_ray_direction.y = ray_direction.y - jitter;
//			jittered_ray_direction.z = ray_direction.z + jitter;
//
//			Ray ray2(jittered_ray_origin, jittered_ray_direction, radiance_ray_type, scene_epsilon);
//			PerRayData_radiance prd2;
//			prd2.importance = 1.f;
//			prd2.depth = 0;
//			prd2.done = false;
//
//			rtTrace(top_object, ray2, prd2);
//
//			/*post_process_output_buffer[launch_index] = make_color(revert_color(input_buffer[launch_index]) + prd2.result);*/
//			post_process_output_buffer[launch_index] = make_color(make_float3(1.0f));
//			additional_rays_count--;
//
//			jitter = static_cast<float>(additional_rays_count) / static_cast<float>(max_per_launch_idx_ray_budget);
//			jitterScale = jitterScale * -1.f;
//			jitter = jitter * jitterScale;
//		}
//	//}
//}

/*--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------*/

RT_PROGRAM void exception()
{
	output_buffer[launch_index] = make_color(bad_color);
}