#include "hip/hip_runtime.h"
/*
* Copyright (c) 2017 NVIDIA CORPORATION. All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions
* are met:
*  * Redistributions of source code must retain the above copyright
*    notice, this list of conditions and the following disclaimer.
*  * Redistributions in binary form must reproduce the above copyright
*    notice, this list of conditions and the following disclaimer in the
*    documentation and/or other materials provided with the distribution.
*  * Neither the name of NVIDIA CORPORATION nor the names of its
*    contributors may be used to endorse or promote products derived
*    from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
* PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
* CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
* EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
* PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
* OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <optixu/optixu_math_namespace.h>
#include "optixPathTracer.h"

#include "random.h"
#include "Buffers.h"
#include "AdaptiveUtil.h"
#include "LevelHoelderAdaptive.h"

using namespace optix;

struct PerRayData_pathtrace
{
	float3 result;
	float3 radiance;
	float3 attenuation;
	float3 origin;
	float3 direction;
	unsigned int seed;
	int depth;
	int countEmitted;
	int done;
	int missed;
	//int isAdaptive;
};

struct PerRayData_pathtrace_shadow
{
	bool inShadow;
};

// Scene wide variables
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(float, far_plane, , );
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

rtDeclareVariable(PerRayData_pathtrace, current_prd, rtPayload, );

rtDeclareVariable(unsigned int, current_level_window_size, , );

RT_PROGRAM void pathtrace_camera_adaptive()
{
	uint2 current_window_centre = get_current_window_centre(launch_index, current_level_window_size);

	output_buffer[launch_index] = output_buffer[current_window_centre];
	output_scene_depth_buffer[launch_index] = output_scene_depth_buffer[current_window_centre];
	output_filter_sum_buffer[launch_index] = output_filter_sum_buffer[current_window_centre];
	output_filter_x_sample_sum_buffer[launch_index] = output_filter_x_sample_sum_buffer[current_window_centre];

	output_current_total_rays_buffer[launch_index] = output_current_total_rays_buffer[current_window_centre];

#ifdef DEBUG_HOELDER
	depth_gradient_buffer[launch_index] = depth_gradient_buffer[current_window_centre];
#endif //DEBUG_HOELDER

	object_ids_buffer[launch_index] = object_ids_buffer[current_window_centre];
}
